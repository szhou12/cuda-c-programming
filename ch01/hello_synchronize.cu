
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void) {
    printf("Hello World from GPU!\n");
}

int main(void)
{
    // hello from cpu
    printf("Hello World from CPU!\n");

    helloFromGPU<<<1, 10>>>();
    hipDeviceSynchronize();

    return 0;
}