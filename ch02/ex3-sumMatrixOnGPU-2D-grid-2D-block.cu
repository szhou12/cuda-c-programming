#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include "../common/common.h"

void initialData(float *ip, int size) 
{
    // generate different seed for random number
    time_t t;
    srand((unsigned int) time(&t));

    for (int i=0; i<size; i++) {
        ip[i] = (int)( rand() & 0xFF ); // Use int for matrix elements
    }
}

void sumMatrixOnHost(int *A, int *B, int *C, const int nx, const int ny)
{
    int *ia = A; // Pointer to the start of matrix A = start index A[0][0]
    int *ib = B;
    int *ic = C;

    for (int iy = 0; iy < ny; iy++) {
        for (int ix = 0; ix < nx; ix++) {
            ic[ix] = ia[ix] + ib[ix];
        }

        ia += nx; // Move to the next row
        ib += nx; // Move to the next row
        ic += nx; // Move to the next row
    }

    return;
}

void checkResult(int *hostRef, int *gpuRef, const int N)
{
    bool match = 1;

    for (int i = 0; i < N; i++) {
        if (hostRef[i] != gpuRef[i]) {
            match = 0
            printf("host %d gpu %d\n", hostRef[i], gpuRef[i]);
            break;
        }
    }

    if (match) {
        printf("Arrays match.\n\n");
    } else {
        printf("Arrays do not match.\n\n");
    }
}

// grid 2D block 2D
__global__ void sumMatrixOnGPU2D(int *MatA, int *MatB, int *MatC, int nx, int ny)
{
    // get matrix index {i, j} from block and thread indices
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    // get global linear index from (i, j) as 2D maktrix is stored in linear 1D format
    unsigned int idx = iy * nx + ix; 

    if (ix < nx && iy < ny) {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of matrix
    int nx = 1 << 14;
    int ny = 1 << 14;

    int nxy = nx * ny;
    int nBytes = nxy * sizeof(int); // array of int
    printf("Matrix size: nx %d ny %d\n", nx, ny);

    // malloc host memory
    // all int type
    int *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (int *)malloc(nBytes);
    h_B = (int *)malloc(nBytes);
    hostRef = (int *)malloc(nBytes);
    gpuRef = (int *)malloc(nBytes);

    // initialize data at host side
    double iStart = seconds();
    initialData(h_A, nxy);
    initialData(h_B, nxy);
    double iElaps = seconds() - iStart;

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // add matrix at host side for result checks
    iStart = seconds();
    sumMatrixOnHost(h_A, h_B, hostRef, nx, ny);
    iElaps = seconds() - iStart;

    // malloc device global memory
    int *d_MatA, *d_MatB, *d_MatC;
    CHECK(hipMalloc((void **)&d_MatA, nBytes));
    CHECK(hipMalloc((void **)&d_MatB, nBytes));
    CHECK(hipMalloc((void **)&d_MatC, nBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice));

    // invoke kernel at host side
    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    iStart = seconds();
    sumMatrixOnGPU2D<<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("sumMatrixOnGPU2D <<<(%d,%d), (%d,%d)>>> elapsed %f sec\n", 
            grid.x, grid.y,
            block.x, block.y, iElaps);
    // check kernel error
    CHECK(hipGetLastError());

    // copy kernel result back to host side
    CHECK(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost));

    // check device results
    checkResult(hostRef, gpuRef, nxy);

    // free device global memory
    CHECK(hipFree(d_MatA));
    CHECK(hipFree(d_MatB));
    CHECK(hipFree(d_MatC));

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    // reset device
    CHECK(hipDeviceReset());

    return (0);

}
