#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""
#include "common.h"

void initialData(float* ip, const int size) {
    int i;

    for (i = 0; i < size; i++) {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

void sumMatrixOnHost(float* A, float* B, float* C, const int nx, const int ny) {
    float* ia = A;
    float* ib = B;
    float* ic = C;

    for (int iy = 0; iy < ny; iy++)
    {
        for (int ix = 0; ix < nx; ix++)
        {
            ic[ix] = ia[ix] + ib[ix];
        }

        ia += nx;
        ib += nx;
        ic += nx;
    }

    return;
}

void checkResult(float* hostRef, float* gpuRef, const int N) {
    double epsilon = 1.0E-8;

    for (int i = 0; i < N; i++) {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
            printf("host %f gpu %f ", hostRef[i], gpuRef[i]);
            printf("*** Arrays do not match. ***\n\n");
            break;
        }
    }
}

// grid 2D block 2D
__global__ void sumMatrixOnGPU2D(float* A, float* B, float* C, int NX, int NY) {
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * NX + ix;

    if (ix < NX && iy < NY) {
        C[idx] = A[idx] + B[idx];
    }
}

int main(int argc, char** argv) {
    printf("sumMatrix program starts ...");

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    CHECK(hipSetDevice(dev));

    // set up data size of matrix
    int nx = 1 << 14;
    int ny = 1 << 14;

    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);

    // malloc host memory
    float* h_A, * h_B, * hostRef, * gpuRef;
    h_A = (float*)malloc(nBytes);
    h_B = (float*)malloc(nBytes);
    hostRef = (float*)malloc(nBytes);
    gpuRef = (float*)malloc(nBytes);

    // init data at host side
    iStart = seconds();
    initialData(h_A, nxy);
    initialData(h_B, nxy);
    iElaps = seconds() - iStart;
    printf("Initialize Matrices on Host: %f ms\n", iElaps * 1000);

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // add matrix at host side for result checks
    iStart = seconds();
    sumMatrixOnHost(h_A, h_B, hostRef, nx, ny);
    iElaps = seconds() - iStart;
    printf("Sum Matrices on Host: %f ms\n", iElaps * 1000);

    // malloc device global memory
    float* d_MatA, * d_MatB, * d_MatC;
    CHECK(hipMalloc((void**)&d_MatA, nBytes));
    CHECK(hipMalloc((void**)&d_MatB, nBytes));
    CHECK(hipMalloc((void**)&d_MatC, nBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice));

    // invoke kernel at host side
    int dimx = 32;
    int dimy = 32;

    if (argc > 2) {
        dimx = atoi(argv[1]);
        dimy = atoi(argv[2]);
    }

    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    // execute the kernel
    CHECK(hipDeviceSynchronize());
    iStart = seconds();
    sumMatrixOnGPU2D <<<grid, block>>> (d_MatA, d_MatB, d_MatC, nx, ny);
    iElaps = seconds() - iStart;
    printf("Sum Matrices on GPU: %f ms\n", iElaps * 1000);

    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    // copy kernel result back to host side
    CHECK(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost));

    // check device results
    iStart = seconds();
    checkResult(hostRef, gpuRef, nxy);
    iElaps = seconds() - iStart;
    printf("Check result on Host: %f ms\n", iElaps * 1000);

    // free device global memory
    CHECK(hipFree(d_MatA));
    CHECK(hipFree(d_MatB));
    CHECK(hipFree(d_MatC));

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    // reset device
    CHECK(hipDeviceReset());

    return EXIT_SUCCESS;






}